#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdint>
#include <stdint.h>
#include <stdio.h>
#include <atomic>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

#include "ggml-cuda.h"
#include "ggml.h"

static_assert(sizeof(half) == sizeof(ggml_fp16_t), "wrong fp16 size");

#define CUDA_CHECK(err)                                                                 \
    do {                                                                                \
        hipError_t err_ = (err);                                                       \
        if (err_ != hipSuccess) {                                                      \
            fprintf(stderr, "CUDA error %d at %s:%d: %s\n", err_, __FILE__, __LINE__,   \
                hipGetErrorString(err_));                                              \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)

#define CUBLAS_CHECK(err)                                                               \
    do {                                                                                \
        hipblasStatus_t err_ = (err);                                                    \
        if (err_ != HIPBLAS_STATUS_SUCCESS) {                                            \
            fprintf(stderr, "cuBLAS error %d at %s:%d\n", err_, __FILE__, __LINE__);    \
            exit(1);                                                                    \
        }                                                                               \
    } while (0)

typedef void (*to_fp32_cuda_t)(const void * x, float * y, int k, hipStream_t stream);

#define QK4_0 32
typedef struct {
    float   d;              // delta
    uint8_t qs[QK4_0 / 2];  // nibbles / quants
} block_q4_0;
static_assert(sizeof(block_q4_0) == sizeof(float) + QK4_0 / 2, "wrong q4_0 block size/padding");

#define QK4_1 32
typedef struct {
    float   d;              // delta
    float   m;              // min
    uint8_t qs[QK4_1 / 2];  // nibbles / quants
} block_q4_1;
static_assert(sizeof(block_q4_1) == sizeof(float) * 2 + QK4_1 / 2, "wrong q4_1 block size/padding");

#define QK4_2 16
typedef struct {
    half  d;                // delta
    uint8_t qs[QK4_2 / 2];  // nibbles / quants
} block_q4_2;
static_assert(sizeof(block_q4_2) == sizeof(ggml_fp16_t) + QK4_2 / 2, "wrong q4_2 block size/padding");

#define QK5_0 32
typedef struct {
    half d;                 // delta
    uint8_t qh[4];          // 5-th bit of quants
    uint8_t qs[QK5_0 / 2];  // nibbles / quants
} block_q5_0;
static_assert(sizeof(block_q5_0) == sizeof(ggml_fp16_t) + sizeof(uint32_t) + QK5_0 / 2, "wrong q5_0 block size/padding");

#define QK5_1 32
typedef struct {
    half d;                 // delta
    half m;                 // min
    uint8_t qh[4];          // 5-th bit of quants
    uint8_t qs[QK5_1 / 2];  // nibbles / quants
} block_q5_1;
static_assert(sizeof(block_q5_1) == 2 * sizeof(ggml_fp16_t) + sizeof(uint32_t) + QK5_1 / 2, "wrong q5_1 block size/padding");

#define QK8_0 32
typedef struct {
    float   d;              // delta
    int8_t  qs[QK8_0];      // quants
} block_q8_0;
static_assert(sizeof(block_q8_0) == sizeof(float) + QK8_0, "wrong q8_0 block size/padding");

static __global__ void dequantize_block_q4_0(const void * vx, float * y) {
    const block_q4_0 * x = (const block_q4_0 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;

    const uint8_t * pp = x[i].qs;

    for (int l = 0; l < QK4_0; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vi0 = vi & 0xf;
        const int8_t vi1 = vi >> 4;

        const float v0 = (vi0 - 8)*d;
        const float v1 = (vi1 - 8)*d;

        y[i*QK4_0 + l + 0] = v0;
        y[i*QK4_0 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q4_1(const void * vx, float * y) {
    const block_q4_1 * x = (const block_q4_1 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;
    const float m = x[i].m;

    const uint8_t * pp = x[i].qs;

    for (int l = 0; l < QK4_1; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vi0 = vi & 0xf;
        const int8_t vi1 = vi >> 4;

        const float v0 = vi0*d + m;
        const float v1 = vi1*d + m;

        y[i*QK4_1 + l + 0] = v0;
        y[i*QK4_1 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q4_2(const void * vx, float * y) {
    const block_q4_2 * x = (const block_q4_2 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;

    const uint8_t * pp = x[i].qs;

    for (int l = 0; l < QK4_2; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vi0 = vi & 0xf;
        const int8_t vi1 = vi >> 4;

        const float v0 = (vi0 - 8)*d;
        const float v1 = (vi1 - 8)*d;

        y[i*QK4_2 + l + 0] = v0;
        y[i*QK4_2 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q5_0(const void * vx, float * y) {
    const block_q5_0 * x = (const block_q5_0 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;

    const uint8_t * pp = x[i].qs;

    uint32_t qh;
    memcpy(&qh, x[i].qh, sizeof(qh));

    for (int l = 0; l < QK5_0; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vh0 = ((qh & (1 << (l + 0))) >> (l + 0)) << 4;
        const int8_t vh1 = ((qh & (1 << (l + 1))) >> (l + 1)) << 4;

        const int8_t vi0 = ((vi & 0xf) | vh0);
        const int8_t vi1 = ((vi >>  4) | vh1);

        const float v0 = (vi0 - 16)*d;
        const float v1 = (vi1 - 16)*d;

        y[i*QK5_0 + l + 0] = v0;
        y[i*QK5_0 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q5_1(const void * vx, float * y) {
    const block_q5_1 * x = (const block_q5_1 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;
    const float m = x[i].m;

    const uint8_t * pp = x[i].qs;

    uint32_t qh;
    memcpy(&qh, x[i].qh, sizeof(qh));

    for (int l = 0; l < QK5_1; l += 2) {
        const uint8_t vi = pp[l/2];

        const int8_t vh0 = ((qh & (1 << (l + 0))) >> (l + 0)) << 4;
        const int8_t vh1 = ((qh & (1 << (l + 1))) >> (l + 1)) << 4;

        const int8_t vi0 = (vi & 0xf) | vh0;
        const int8_t vi1 = (vi >>  4) | vh1;

        const float v0 = vi0*d + m;
        const float v1 = vi1*d + m;

        y[i*QK5_1 + l + 0] = v0;
        y[i*QK5_1 + l + 1] = v1;
    }
}

static __global__ void dequantize_block_q8_0(const void * vx, float * y) {
    const block_q8_0 * x = (const block_q8_0 *) vx;

    const int i = blockIdx.x;

    const float d = x[i].d;

    const int8_t * pp = x[i].qs;

    for (int l = 0; l < QK8_0; l++) {
        const int8_t vi = pp[l];

        y[i*QK8_0 + l] = vi*d;
    }
}

static void dequantize_row_q4_0_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK4_0;
    dequantize_block_q4_0<<<nb, 1, 0, stream>>>(vx, y);
}

static void dequantize_row_q4_1_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK4_1;
    dequantize_block_q4_1<<<nb, 1, 0, stream>>>(vx, y);
}

static void dequantize_row_q4_2_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK4_2;
    dequantize_block_q4_2<<<nb, 1, 0, stream>>>(vx, y);
}

static void dequantize_row_q5_0_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK5_0;
    dequantize_block_q5_0<<<nb, 1, 0, stream>>>(vx, y);
}

static void dequantize_row_q5_1_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK5_1;
    dequantize_block_q5_1<<<nb, 1, 0, stream>>>(vx, y);
}

static void dequantize_row_q8_0_cuda(const void * vx, float * y, int k, hipStream_t stream) {
    const int nb = k / QK8_0;
    dequantize_block_q8_0<<<nb, 1, 0, stream>>>(vx, y);
}

// TODO: optimize
static __global__ void convert_fp16_to_fp32(const void * vx, float * y) {
    const half * x = (const half *) vx;

    const int i = blockIdx.x;

    y[i] = __half2float(x[i]);
}

static void convert_fp16_to_fp32_cuda(const void * x, float * y, int k, hipStream_t stream) {
    convert_fp16_to_fp32<<<k, 1, 0, stream>>>(x, y);
}

static to_fp32_cuda_t ggml_get_to_fp32_cuda(ggml_type type) {
    switch (type) {
        case GGML_TYPE_Q4_0:
            return dequantize_row_q4_0_cuda;
        case GGML_TYPE_Q4_1:
            return dequantize_row_q4_1_cuda;
        case GGML_TYPE_Q4_2:
            return dequantize_row_q4_2_cuda;
        case GGML_TYPE_Q5_0:
            return dequantize_row_q5_0_cuda;
        case GGML_TYPE_Q5_1:
            return dequantize_row_q5_1_cuda;
        case GGML_TYPE_Q8_0:
            return dequantize_row_q8_0_cuda;
        case GGML_TYPE_F16:
            return convert_fp16_to_fp32_cuda;
        default:
            return nullptr;
    }
}

// buffer pool for cuda
#define MAX_CUDA_BUFFERS 16

struct scoped_spin_lock {
    std::atomic_flag& lock;
    scoped_spin_lock(std::atomic_flag& lock) : lock(lock) {
        while (lock.test_and_set(std::memory_order_acquire)) {
            ; // spin
        }
    }
    ~scoped_spin_lock() {
        lock.clear(std::memory_order_release);
    }
    scoped_spin_lock(const scoped_spin_lock&) = delete;
    scoped_spin_lock& operator=(const scoped_spin_lock&) = delete;
};

struct cuda_buffer {
    void * ptr = nullptr;
    size_t size = 0;
};

static cuda_buffer g_cuda_buffer_pool[MAX_CUDA_BUFFERS];
static std::atomic_flag g_cuda_pool_lock = ATOMIC_FLAG_INIT;

static void * ggml_cuda_pool_malloc(size_t size, size_t * actual_size) {
    scoped_spin_lock lock(g_cuda_pool_lock);

    for (int i = 0; i < MAX_CUDA_BUFFERS; ++i) {
        cuda_buffer& b = g_cuda_buffer_pool[i];
        if (b.size >= size && b.ptr != nullptr) {
            void * ptr = b.ptr;
            *actual_size = b.size;
            b.ptr = nullptr;
            b.size = 0;
            return ptr;
        }
    }
    void * ptr;
    CUDA_CHECK(hipMalloc((void **) &ptr, size));
    *actual_size = size;
    return ptr;
}

static void ggml_cuda_pool_free(void * ptr, size_t size) {
    scoped_spin_lock lock(g_cuda_pool_lock);

    for (int i = 0; i < MAX_CUDA_BUFFERS; ++i) {
        cuda_buffer& b = g_cuda_buffer_pool[i];
        if (b.ptr == nullptr) {
            b.ptr = ptr;
            b.size = size;
            return;
        }
    }
    fprintf(stderr, "WARNING: cuda buffer pool full, increase MAX_CUDA_BUFFERS\n");
    CUDA_CHECK(hipFree(ptr));
}

#define GGML_CUDA_MAX_STREAMS 8
#define GGML_CUDA_MAX_EVENTS 64
static hipblasHandle_t g_cublasH = nullptr;
static hipStream_t g_cudaStreams[GGML_CUDA_MAX_STREAMS] = { nullptr };
static hipStream_t g_cudaStreams2[GGML_CUDA_MAX_STREAMS] = { nullptr };
static hipEvent_t g_cudaEvents[GGML_CUDA_MAX_EVENTS] = { nullptr };

void ggml_init_cublas() {
    if (g_cublasH == nullptr) {
        // create streams
        for (int i = 0; i < GGML_CUDA_MAX_STREAMS; ++i) {
            CUDA_CHECK(hipStreamCreateWithFlags(&g_cudaStreams[i], hipStreamNonBlocking));
            CUDA_CHECK(hipStreamCreateWithFlags(&g_cudaStreams2[i], hipStreamNonBlocking));
        }
        // create events
        for (int i = 0; i < GGML_CUDA_MAX_EVENTS; ++i) {
            CUDA_CHECK(hipEventCreateWithFlags(&g_cudaEvents[i], hipEventDisableTiming));
        }

        // create cublas handle
        CUBLAS_CHECK(hipblasCreate(&g_cublasH));
        CUBLAS_CHECK(hipblasSetMathMode(g_cublasH, HIPBLAS_TF32_TENSOR_OP_MATH));

        // configure logging to stdout
        // CUBLAS_CHECK(cublasLoggerConfigure(1, 1, 0, nullptr));
    }
}

void * ggml_cuda_host_malloc(size_t size) {
    if (getenv("GGML_CUDA_NO_PINNED") != nullptr) {
        return nullptr;
    }

    void * ptr = nullptr;
    hipError_t err = hipHostMalloc((void **) &ptr, size);
    if (err != hipSuccess) {
        fprintf(stderr, "WARNING: failed to allocate %.2f MB of pinned memory: %s\n",
            size/1024.0/1024.0, hipGetErrorString(err));
        return nullptr;
    }

    return ptr;
}

void ggml_cuda_host_free(void * ptr) {
    CUDA_CHECK(hipHostFree(ptr));
}

static hipError_t ggml_cuda_h2d_tensor_2d(void * dst, const struct ggml_tensor * src, uint64_t i3, uint64_t i2, hipStream_t stream) {
    const uint64_t ne0 = src->ne[0];
    const uint64_t ne1 = src->ne[1];
    const uint64_t nb0 = src->nb[0];
    const uint64_t nb1 = src->nb[1];
    const uint64_t nb2 = src->nb[2];
    const uint64_t nb3 = src->nb[3];
    const enum ggml_type type = src->type;
    const size_t ts = ggml_type_size(type);
    const size_t bs = ggml_blck_size(type);

    const void * x = (const void *) ((const char *) src->data + i2*nb2 + i3*nb3);
    if (nb0 == ts && nb1 == ts*ne0/bs) {
        return hipMemcpyAsync(dst, x, ne1*nb1, hipMemcpyHostToDevice, stream);
    } else if (nb0 == ts) {
        return hipMemcpy2DAsync(dst, ts*ne0/bs, x, nb1, ts*ne0/bs, ne1, hipMemcpyHostToDevice, stream);
    } else {
        for (uint64_t i1 = 0; i1 < ne1; i1++) {
            const void * rx = (const void *) ((const char *) x + i1*nb1);
            void * rd = (void *) ((char *) dst + i1*ts*ne0/bs);
            // pretend the row is a matrix with cols=1
            hipError_t r = hipMemcpy2DAsync(rd, ts/bs, rx, nb0, ts/bs, ne0, hipMemcpyHostToDevice, stream);
            if (r != hipSuccess) return r;
        }
        return hipSuccess;
    }
}

static void ggml_cuda_mul_mat_f32(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];
    const int64_t ne03 = src0->ne[3];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];

    const int nb2  = dst->nb[2];
    const int nb3  = dst->nb[3];

    const float alpha = 1.0f;
    const float beta = 0.0f;
    const int x_ne = ne01 * ne00;
    const int y_ne = ne11 * ne10;
    const int d_ne = ne11 * ne01;
    const int n_mm = ne03 * ne02;

    size_t x_size, y_size, d_size;
    float * d_X = (float *) ggml_cuda_pool_malloc(n_mm * sizeof(float) * x_ne, &x_size);
    float * d_Y = (float *) ggml_cuda_pool_malloc(n_mm * sizeof(float) * y_ne, &y_size);
    float * d_D = (float *) ggml_cuda_pool_malloc(n_mm * sizeof(float) * d_ne, &d_size);

    for (int64_t i03 = 0; i03 < ne03; i03++) {
        for (int64_t i02 = 0; i02 < ne02; i02++) {
            int i = i03*ne02 + i02;
            hipStream_t cudaStream = g_cudaStreams[i % GGML_CUDA_MAX_STREAMS];

            float * c_X = d_X + i * x_ne;
            float * c_Y = d_Y + i * y_ne;
            float * c_D = d_D + i * d_ne;

            // copy data to device
            CUDA_CHECK(ggml_cuda_h2d_tensor_2d(c_X, src0, i03, i02, cudaStream));
            CUDA_CHECK(ggml_cuda_h2d_tensor_2d(c_Y, src1, i03, i02, cudaStream));

            // compute
            CUBLAS_CHECK(hipblasSetStream(g_cublasH, cudaStream));
            CUBLAS_CHECK(
                hipblasSgemm(g_cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        ne01, ne11, ne10,
                        &alpha, c_X, ne00,
                                c_Y, ne10,
                        &beta,  c_D, ne01));

            // copy dst to host
            float * d = (float *) ((char *) dst->data + i02*nb2 + i03*nb3);
            CUDA_CHECK(hipMemcpyAsync(d, c_D, sizeof(float) * d_ne, hipMemcpyDeviceToHost, cudaStream));
        }
    }

    CUDA_CHECK(hipDeviceSynchronize());
    ggml_cuda_pool_free(d_X, x_size);
    ggml_cuda_pool_free(d_Y, y_size);
    ggml_cuda_pool_free(d_D, d_size);
}

static void ggml_cuda_mul_mat_f16(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, void * wdata, size_t /* wsize */) {
    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];
    const int64_t ne03 = src0->ne[3];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];

    const int nb10 = src1->nb[0];
    const int nb11 = src1->nb[1];
    const int nb12 = src1->nb[2];
    const int nb13 = src1->nb[3];

    const int nb2  = dst->nb[2];
    const int nb3  = dst->nb[3];

    const float alpha = 1.0f;
    const float beta = 0.0f;
    const int x_ne = ne01 * ne00;
    const int y_ne = ne11 * ne10;
    const int d_ne = ne11 * ne01;
    const int n_mm = ne03 * ne02;

    size_t x_size, y_size, d_size;
    half  * d_X =  (half *) ggml_cuda_pool_malloc(n_mm * sizeof(half) * x_ne, &x_size);
    half  * d_Y =  (half *) ggml_cuda_pool_malloc(n_mm * sizeof(half) * y_ne, &y_size);
    float * d_D = (float *) ggml_cuda_pool_malloc(n_mm * sizeof(float) * d_ne, &d_size);

    bool src1_cont_rows = nb10 == sizeof(float);
    bool src1_cont_cols = (size_t)nb11 == ne11*sizeof(float);

    for (int64_t i03 = 0; i03 < ne03; i03++) {
        for (int64_t i02 = 0; i02 < ne02; i02++) {
            int i = i03*ne02 + i02;
            hipStream_t cudaStream = g_cudaStreams[i % GGML_CUDA_MAX_STREAMS];

            half  * c_X = d_X + i * x_ne;
            half  * c_Y = d_Y + i * y_ne;
            float * c_D = d_D + i * d_ne;

            // copy src0 to device
            CUDA_CHECK(ggml_cuda_h2d_tensor_2d(c_X, src0, i03, i02, cudaStream));

            // convert src1 to fp16
            // TODO: use multiple threads
            ggml_fp16_t * const tmp = (ggml_fp16_t *) wdata + (ne11 * ne10) * (i03 * ne02 + i02);
            char * src1i = (char *) src1->data + i03*nb13 + i02*nb12;
            if (src1_cont_rows) {
                if (src1_cont_cols) {
                    ggml_fp32_to_fp16_row((float *) src1i, tmp, ne10*ne11);
                }
                else {
                    for (int64_t i01 = 0; i01 < ne11; i01++) {
                        ggml_fp32_to_fp16_row((float *) (src1i + i01*nb11), tmp + i01*ne10, ne10);
                    }
                }
            }
            else {
                for (int64_t i01 = 0; i01 < ne11; i01++) {
                    for (int64_t i00 = 0; i00 < ne10; i00++) {
                        // very slow due to no inlining
                        tmp[i01*ne10 + i00] = ggml_fp32_to_fp16(*(float *) (src1i + i01*nb11 + i00*nb10));
                    }
                }
            }

            // copy src1 to device
            CUDA_CHECK(hipMemcpyAsync(c_Y, tmp, sizeof(half) * y_ne, hipMemcpyHostToDevice, cudaStream));

            // compute
            CUBLAS_CHECK(hipblasSetStream(g_cublasH, cudaStream));
            CUBLAS_CHECK(
                hipblasGemmEx(g_cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        ne01, ne11, ne10,
                        &alpha, c_X, HIP_R_16F, ne00,
                                c_Y, HIP_R_16F, ne10,
                        &beta,  c_D, HIP_R_32F, ne01,
                        HIPBLAS_COMPUTE_32F_FAST_16F,
                        HIPBLAS_GEMM_DEFAULT));

            // copy dst to host
            float * d = (float *) ((char *) dst->data + i02*nb2 + i03*nb3);
            CUDA_CHECK(hipMemcpyAsync(d, c_D, sizeof(float) * d_ne, hipMemcpyDeviceToHost, cudaStream));
        }
    }

    CUDA_CHECK(hipDeviceSynchronize());
    ggml_cuda_pool_free(d_X, x_size);
    ggml_cuda_pool_free(d_Y, y_size);
    ggml_cuda_pool_free(d_D, d_size);
}

static void ggml_cuda_mul_mat_q_f32(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst) {
    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];
    const int64_t ne03 = src0->ne[3];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];

    const int nb2  = dst->nb[2];
    const int nb3  = dst->nb[3];
    const ggml_type type = src0->type;

    const float alpha = 1.0f;
    const float beta = 0.0f;
    const int x_ne = ne01 * ne00;
    const int y_ne = ne11 * ne10;
    const int d_ne = ne11 * ne01;
    const int n_mm = ne03 * ne02;
    const size_t q_sz = ggml_type_size(type) * x_ne / ggml_blck_size(type);

    size_t x_size, y_size, d_size, q_size;
    float * d_X = (float *) ggml_cuda_pool_malloc(n_mm * sizeof(float) * x_ne, &x_size);
    float * d_Y = (float *) ggml_cuda_pool_malloc(n_mm * sizeof(float) * y_ne, &y_size);
    float * d_D = (float *) ggml_cuda_pool_malloc(n_mm * sizeof(float) * d_ne, &d_size);
    char  * d_Q = (char  *) ggml_cuda_pool_malloc(n_mm * q_sz, &q_size);

    const to_fp32_cuda_t to_fp32_cuda = ggml_get_to_fp32_cuda(type);
    GGML_ASSERT(to_fp32_cuda != nullptr);

    for (int64_t i03 = 0; i03 < ne03; i03++) {
        for (int64_t i02 = 0; i02 < ne02; i02++) {
            int i = i03*ne02 + i02;
            hipStream_t cudaStream = g_cudaStreams[i % GGML_CUDA_MAX_STREAMS];
            hipStream_t cudaStream2 = g_cudaStreams2[i % GGML_CUDA_MAX_STREAMS];
            hipEvent_t  cudaEvent = g_cudaEvents[i % GGML_CUDA_MAX_EVENTS];

            float * c_X = d_X + i * x_ne;
            float * c_Y = d_Y + i * y_ne;
            float * c_D = d_D + i * d_ne;
            char  * c_Q = d_Q + i * q_sz;

            // copy src0 and convert to fp32 on device
            CUDA_CHECK(ggml_cuda_h2d_tensor_2d(c_Q, src0, i03, i02, cudaStream2));
            to_fp32_cuda(c_Q, c_X, x_ne, cudaStream2);
            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipEventRecord(cudaEvent, cudaStream2));

            // copy src1 to device
            CUDA_CHECK(ggml_cuda_h2d_tensor_2d(c_Y, src1, i03, i02, cudaStream));

            // wait for conversion
            CUDA_CHECK(hipStreamWaitEvent(cudaStream, cudaEvent, 0));

            // compute
            CUBLAS_CHECK(hipblasSetStream(g_cublasH, cudaStream));
            CUBLAS_CHECK(
                hipblasSgemm(g_cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        ne01, ne11, ne10,
                        &alpha, c_X, ne00,
                                c_Y, ne10,
                        &beta,  c_D, ne01));

            // copy dst to host
            float * d = (float *) ((char *) dst->data + i02*nb2 + i03*nb3);
            CUDA_CHECK(hipMemcpyAsync(d, c_D, sizeof(float) * d_ne, hipMemcpyDeviceToHost, cudaStream));
        }
    }

    CUDA_CHECK(hipDeviceSynchronize());
    ggml_cuda_pool_free(d_X, x_size);
    ggml_cuda_pool_free(d_Y, y_size);
    ggml_cuda_pool_free(d_D, d_size);
    ggml_cuda_pool_free(d_Q, q_size);
}

bool ggml_cuda_can_mul_mat(const struct ggml_tensor * src0, const struct ggml_tensor * src1, struct ggml_tensor * dst) {
    const int64_t ne10 = src1->ne[0];

    const int64_t ne0 = dst->ne[0];
    const int64_t ne1 = dst->ne[1];

    // TODO: find the optimal values for these
    if ((src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16 || ggml_is_quantized(src0->type)) &&
        src1->type == GGML_TYPE_F32 &&
        dst->type == GGML_TYPE_F32 &&
        (ne0 >= 32 && ne1 >= 32 && ne10 >= 32)) {

        return true;
    }

    return false;
}

bool ggml_cuda_mul_mat_use_f16(const struct ggml_tensor * src0, const struct ggml_tensor * src1, struct ggml_tensor * /* dst */) {
    size_t src0_sz = ggml_nbytes(src0);
    size_t src1_sz = ggml_nbytes(src1);

    // mul_mat_q: src0 is converted to fp32 on device
    size_t mul_mat_q_transfer = src0_sz + src1_sz;

    // mul_mat_f16: src1 is converted to fp16 on cpu
    size_t mul_mat_f16_transfer = src0_sz + sizeof(half) * ggml_nelements(src1);

    // choose the smaller one to transfer to the device
    // TODO: this is not always the best choice due to the overhead of converting to fp16
    return mul_mat_f16_transfer < mul_mat_q_transfer;
}

void ggml_cuda_mul_mat(const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, void * wdata, size_t wsize) {
    GGML_ASSERT(ggml_cuda_can_mul_mat(src0, src1, dst));

    if (src0->type == GGML_TYPE_F32) {
        ggml_cuda_mul_mat_f32(src0, src1, dst);
    }
    else if (src0->type == GGML_TYPE_F16) {
        if (ggml_cuda_mul_mat_use_f16(src0, src1, dst)) {
            ggml_cuda_mul_mat_f16(src0, src1, dst, wdata, wsize);
        }
        else {
            ggml_cuda_mul_mat_q_f32(src0, src1, dst);
        }
    }
    else if (ggml_is_quantized(src0->type)) {
        ggml_cuda_mul_mat_q_f32(src0, src1, dst);
    }
    else {
        GGML_ASSERT(false);
    }
}

size_t ggml_cuda_mul_mat_get_wsize(const struct ggml_tensor * src0, const struct ggml_tensor * src1, struct ggml_tensor * dst) {
    if (ggml_cuda_mul_mat_use_f16(src0, src1, dst)) {
        return ggml_nelements(src1) * sizeof(ggml_fp16_t);
    }
    else {
        return 0;
    }
}
